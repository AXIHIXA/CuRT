#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <glm/glm.hpp>
#include <opencv2/opencv.hpp>
#include <thrust/device_vector.h>
#include <thrust/host_vector.h>

#include "ch03/Ray.cuh"
#include "util/CudaUtil.h"


constexpr dim3 kBlockDim {32U, 32U, 1U};

constexpr int kWidth = 1200;

constexpr int kHeight = 600;


__device__
glm::vec3 color(const Ray & r)
{
    float t = 0.5f * (r.d().y + 1.0f);
    return (1.0f - t) * glm::vec3(1.0f) + t * glm::vec3(0.5f, 0.7f, 1.0f);
}


__global__
void render(
        glm::vec3 * __restrict__ fb,
        int width,
        int height,
        glm::vec3 lowerLeft,
        glm::vec3 horizontal,
        glm::vec3 vertical,
        glm::vec3 origin
)
{
    auto x = static_cast<int>(blockIdx.x * blockDim.x + threadIdx.x);
    auto y = static_cast<int>(blockIdx.y * blockDim.y + threadIdx.y);

    if (x < width and y < height)
    {
        float u = static_cast<float>(x) / static_cast<float>(width);
        float v = static_cast<float>(y) / static_cast<float>(height);

        // Ray from global-space origin through viewport pixel
        Ray r(origin, lowerLeft + u * horizontal + v * vertical);
        fb[y * width + x] = color(r);
    }
}


int main(int argc, char * argv[])
{
    thrust::device_vector<glm::vec3> dFb(kWidth * kHeight, {1.0f, 1.0f, 1.0f});

    dim3 mGridDim = {
            (kWidth + kBlockDim.x - 1U) / kBlockDim.x,
            (kHeight + kBlockDim.y - 1U) / kBlockDim.y,
            1U
    };

    render<<<mGridDim, kBlockDim>>>(
            dFb.data().get(),
            kWidth,
            kHeight,
            glm::vec3(-2.0f, -1.0f, -1.0f),
            glm::vec3(4.0f, 0.0f, 0.0f),
            glm::vec3(0.0f, 2.0f, 0.0f),
            glm::vec3(0.0f, 0.0f, 0.0f)
    );
    CUDA_CHECK_LAST_ERROR();
    CUDA_CHECK(hipDeviceSynchronize());

    thrust::host_vector<glm::vec3> hFb = dFb;
    cv::Mat img(kHeight, kWidth, CV_32FC3, hFb.data());
    cv::cvtColor(img, img, cv::COLOR_RGB2BGR);
    cv::flip(img, img, 0);
    cv::imshow("Chap 03 Ray", img);
    cv::waitKey();

    return EXIT_SUCCESS;
}
